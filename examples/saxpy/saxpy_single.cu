#include "hip/hip_runtime.h"
/*
 * Copyright 2015 Stanford University and NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdlib.h>

#include "cudaDMA.h"
#include "params.h"

/*
 * This version of saxpy uses cudaDMA for DMAs (but requires 2 CTAs/SM) for double buffering.
 */
__global__ void saxpy_cudaDMA ( float* y, float* x, float a, clock_t * timer_vals) 
{
  __shared__ float sdata_x0 [COMPUTE_THREADS_PER_CTA];
  __shared__ float sdata_y0 [COMPUTE_THREADS_PER_CTA];

  cudaDMASequential<true, 16, DMA_SZ, DMA_THREADS_PER_LD>
    dma_ld_x_0 (1, COMPUTE_THREADS_PER_CTA, COMPUTE_THREADS_PER_CTA );
  cudaDMASequential<true, 16, DMA_SZ, DMA_THREADS_PER_LD>
    dma_ld_y_0 (2, COMPUTE_THREADS_PER_CTA, COMPUTE_THREADS_PER_CTA + DMA_THREADS_PER_LD );

  int tid = threadIdx.x ;

  if ( tid < COMPUTE_THREADS_PER_CTA ) {
    unsigned int idx;
    int i;
    float tmp_x;
    float tmp_y;
    
    // Preamble:
    dma_ld_x_0.start_async_dma();
    dma_ld_y_0.start_async_dma();
    #pragma unroll 1
    for (i = 0; i < NUM_ITERS-1; ++i) {
      dma_ld_x_0.wait_for_dma_finish();
      tmp_x = sdata_x0[tid];
      dma_ld_x_0.start_async_dma();
      dma_ld_y_0.wait_for_dma_finish();
      tmp_y = sdata_y0[tid];
      dma_ld_y_0.start_async_dma();
      idx = i * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + threadIdx.x;
      y[idx] = a * tmp_x + tmp_y;
    }
    // Postamble:
    dma_ld_x_0.wait_for_dma_finish();
    tmp_x = sdata_x0[tid];
    dma_ld_y_0.wait_for_dma_finish();
    tmp_y = sdata_y0[tid];
    idx = i * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + threadIdx.x;
    y[idx] = a * tmp_x + tmp_y;

  } else if (dma_ld_x_0.owns_this_thread()) {
    #pragma unroll 1
    for (unsigned int j = 0; j < NUM_ITERS; ++j) {
      // idx is a pointer to the base of the chunk of memory to copy
      unsigned int idx = j * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA;
      dma_ld_x_0.execute_dma( &x[idx], sdata_x0 );
    }
  } else if (dma_ld_y_0.owns_this_thread()) {
    #pragma unroll 1
    for (unsigned int j = 0; j < NUM_ITERS; ++j) {
      unsigned int idx = j * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA;
      dma_ld_y_0.execute_dma( &y[idx], sdata_y0 );
    }
  }
}



