#include "hip/hip_runtime.h"
/*
 * Copyright 2015 Stanford University and NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdlib.h>

#include "cudaDMA.h"
#include "params.h"

/*
 * This version of saxpy uses cudaDMA for DMAs with manual double buffering.
 */
__global__ void saxpy_cudaDMA_doublebuffer ( float* y, float* x, float a, clock_t * timer_vals) 
{
  __shared__ float sdata_x0 [COMPUTE_THREADS_PER_CTA];
  __shared__ float sdata_x1 [COMPUTE_THREADS_PER_CTA];
  __shared__ float sdata_y0 [COMPUTE_THREADS_PER_CTA];
  __shared__ float sdata_y1 [COMPUTE_THREADS_PER_CTA];

  cudaDMASequential<true, 16, DMA_SZ, DMA_THREADS_PER_LD>
    dma_ld_x_0 (1, COMPUTE_THREADS_PER_CTA, COMPUTE_THREADS_PER_CTA);
  cudaDMASequential<true, 16, DMA_SZ, DMA_THREADS_PER_LD>
    dma_ld_y_0 (2, COMPUTE_THREADS_PER_CTA, COMPUTE_THREADS_PER_CTA + DMA_THREADS_PER_LD);
  cudaDMASequential<true, 16, DMA_SZ, DMA_THREADS_PER_LD>
    dma_ld_x_1 (3, COMPUTE_THREADS_PER_CTA, COMPUTE_THREADS_PER_CTA + 2*DMA_THREADS_PER_LD);
  cudaDMASequential<true, 16, DMA_SZ, DMA_THREADS_PER_LD>
    dma_ld_y_1 (4, COMPUTE_THREADS_PER_CTA, COMPUTE_THREADS_PER_CTA + 3*DMA_THREADS_PER_LD);

  int tid = threadIdx.x ;

  if ( tid < COMPUTE_THREADS_PER_CTA ) {
    unsigned int idx;
    int i;
    float tmp_x;
    float tmp_y;
    
    // Preamble:
    dma_ld_x_0.start_async_dma();
    dma_ld_y_0.start_async_dma();
    dma_ld_x_1.start_async_dma();
    dma_ld_y_1.start_async_dma();
    #pragma unroll 1
    for (i = 0; i < NUM_ITERS-2; i += 2) {
      
      // Phase 1:
      dma_ld_x_0.wait_for_dma_finish();
      tmp_x = sdata_x0[tid];
      dma_ld_x_0.start_async_dma();
      dma_ld_y_0.wait_for_dma_finish();
      tmp_y = sdata_y0[tid];
      dma_ld_y_0.start_async_dma();
      idx = i * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + threadIdx.x;
      y[idx] = a * tmp_x + tmp_y;

      // Phase 2:
      dma_ld_x_1.wait_for_dma_finish();
      tmp_x = sdata_x1[tid];
      dma_ld_x_1.start_async_dma();
      dma_ld_y_1.wait_for_dma_finish();
      tmp_y = sdata_y1[tid];
      dma_ld_y_1.start_async_dma();
      idx = (i+1) * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + threadIdx.x;
      y[idx] = a * tmp_x + tmp_y;
    }
      
    // Postamble
    dma_ld_x_0.wait_for_dma_finish();
    tmp_x = sdata_x0[tid];
    dma_ld_y_0.wait_for_dma_finish();
    tmp_y = sdata_y0[tid];
    idx = i * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + threadIdx.x;
    y[idx] = a * tmp_x + tmp_y;
    dma_ld_x_1.wait_for_dma_finish();
    tmp_x = sdata_x1[tid];
    dma_ld_y_1.wait_for_dma_finish();
    tmp_y = sdata_y1[tid];
    idx = (i+1) * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + threadIdx.x;
    y[idx] = a * tmp_x + tmp_y;

  } else if (dma_ld_x_0.owns_this_thread()) {
    #pragma unroll 1
    for (unsigned int j = 0; j < NUM_ITERS; j+=2) {
      // idx is a pointer to the base of the chunk of memory to copy
      unsigned int idx = j * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA;
      dma_ld_x_0.execute_dma( &x[idx], sdata_x0 );
    }
  } else if (dma_ld_y_0.owns_this_thread()) {
    #pragma unroll 1
    for (unsigned int j = 0; j < NUM_ITERS; j+=2) {
      unsigned int idx = j * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA;
      dma_ld_y_0.execute_dma( &y[idx], sdata_y0 );
    }
  } else if (dma_ld_x_1.owns_this_thread()) {
    #pragma unroll 1
    for (unsigned int j = 1; j < NUM_ITERS; j+=2) {
      unsigned int idx = j * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA;
      dma_ld_x_1.execute_dma( &x[idx], sdata_x1 );
    }
  } else if (dma_ld_y_1.owns_this_thread()) {
    #pragma unroll 1
    for (unsigned int j = 1; j < NUM_ITERS; j+=2) {
      unsigned int idx = j * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA;
      dma_ld_y_1.execute_dma( &y[idx], sdata_y1 );
    }
  }
}

