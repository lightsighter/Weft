#include "hip/hip_runtime.h"
/*
 * Copyright 2015 Stanford University and NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <cmath>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "cudaDMA.h"

#define SIZE_N	        896
#define SIZE_M		SIZE_N

#define DMA_KERNEL			sgemvn_cuda_dma_vec_single
#define COMPUTE_THREADS_PER_CTA		128	
#define DMA_THREADS_PER_LD		32	
#define DMA_LDS				1
#ifndef VEC_ELMTS
#define VEC_ELMTS		 	128	
#endif

#ifndef SGEMV_ITERS
#define SGEMV_ITERS                     128
#endif

__global__ void
__launch_bounds__(160,1)
sgemvn_cuda_dma_vec_single(int n, int m, int n1, float alpha, float *A, int lda, float *x, float *y)
{
	__shared__ float buff[VEC_ELMTS];

	cudaDMASequential<true,16,4*VEC_ELMTS,DMA_THREADS_PER_LD>
	  dma_ld_0(1,COMPUTE_THREADS_PER_CTA,COMPUTE_THREADS_PER_CTA);

	if (threadIdx.x < COMPUTE_THREADS_PER_CTA)
	{
		dma_ld_0.start_async_dma();	
		int ind = blockIdx.x*COMPUTE_THREADS_PER_CTA + threadIdx.x;

		A += ind;

		float res = 0.f;

#ifdef DYNAMIC
                #pragma unroll 1
		for(int i=0; i<n1; i += VEC_ELMTS)
#else
                for(int i=0; i<SGEMV_ITERS; i++)
#endif
		{
			dma_ld_0.wait_for_dma_finish();
			#pragma unroll
			for(int j=0; j < VEC_ELMTS; j++)
			{
				res+=A[0]*buff[j];
				A+=lda;
			}
			dma_ld_0.start_async_dma();
		}

		#if 0
		if (m>n1)
		{
			buff[threadIdx.x]  = x[n1];

			__syncthreads();
			for(int j=0; j<(m-n1); j++)
			{
				 res += A[0]*buff[j];
				 A+=lda;
			}
		  }
		#endif

		if (ind<n)
			y[ind] = alpha * res;
	}
	else if (dma_ld_0.owns_this_thread())
	{
#ifdef DYNAMIC
                #pragma unroll 1
		for (int idx=0; idx<n1; idx += VEC_ELMTS)
#else
                for (int idx=0; idx<SGEMV_ITERS; idx++)
#endif
		{
			dma_ld_0.execute_dma(x,buff);
			x += VEC_ELMTS;
		}	
		dma_ld_0.wait_for_dma_start();
	}
}

