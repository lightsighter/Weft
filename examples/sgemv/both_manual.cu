#include "hip/hip_runtime.h"
/*
 * Copyright 2015 Stanford University and NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <cmath>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "cudaDMA.h"

#define SIZE_N	        896
#define SIZE_M		SIZE_N

#define DMA_KERNEL			sgemvn_cuda_dma_both_manual
#define COMPUTE_THREADS_PER_CTA		128
#define DMA_THREADS_PER_LD		32
#define DMA_LDS				9	
#ifndef VEC_ELMTS
#define VEC_ELMTS			32	
#endif

#ifndef SGEMV_ITERS
#define SGEMV_ITERS                     64
#endif

__global__ void
__launch_bounds__(416,1)
sgemvn_cuda_dma_both_manual(int n, int m, int n1, float alpha, float *A, int lda, float *x, float *y)
{
	__shared__ float buff0[VEC_ELMTS];
	__shared__ float buff1[VEC_ELMTS];
	__shared__ float mat0[VEC_ELMTS][COMPUTE_THREADS_PER_CTA];	
	__shared__ float mat1[VEC_ELMTS][COMPUTE_THREADS_PER_CTA];

	cudaDMASequential<true,16,4*VEC_ELMTS,DMA_THREADS_PER_LD>
	  dma_ld_0(1,COMPUTE_THREADS_PER_CTA,COMPUTE_THREADS_PER_CTA);

	cudaDMASequential<true,16,4*VEC_ELMTS,DMA_THREADS_PER_LD>
	  dma_ld_1(2,COMPUTE_THREADS_PER_CTA,COMPUTE_THREADS_PER_CTA);

	cudaDMAStrided<true,16,4*COMPUTE_THREADS_PER_CTA,8*DMA_THREADS_PER_LD,VEC_ELMTS>
	  dma_ld_2(3,COMPUTE_THREADS_PER_CTA,COMPUTE_THREADS_PER_CTA+1*DMA_THREADS_PER_LD,4*lda);

	cudaDMAStrided<true,16,4*COMPUTE_THREADS_PER_CTA,8*DMA_THREADS_PER_LD,VEC_ELMTS>
	  dma_ld_3(4,COMPUTE_THREADS_PER_CTA,COMPUTE_THREADS_PER_CTA+1*DMA_THREADS_PER_LD,4*lda);

	if (threadIdx.x < COMPUTE_THREADS_PER_CTA)
	{
		dma_ld_0.start_async_dma();	
		dma_ld_1.start_async_dma();
		dma_ld_2.start_async_dma();
		dma_ld_3.start_async_dma();

		float res = 0.f;

#ifdef DYNAMIC
                #pragma unroll 1
		for(int i=0; i<n1; i += (VEC_ELMTS*2))
#else
                for(int i=0; i<SGEMV_ITERS; i++)
#endif
		{
			dma_ld_0.wait_for_dma_finish();
			dma_ld_2.wait_for_dma_finish();
			#pragma unroll
			for(int j=0; j < VEC_ELMTS; j++)
			{
				res+=mat0[j][threadIdx.x]*buff0[j];
			}
			dma_ld_0.start_async_dma();
			dma_ld_2.start_async_dma();

			dma_ld_1.wait_for_dma_finish();
			dma_ld_3.wait_for_dma_finish();
			#pragma unroll
			for (int j=0; j < VEC_ELMTS; j++)
			{
				res+=mat1[j][threadIdx.x]*buff1[j];
			}
			dma_ld_1.start_async_dma();
			dma_ld_3.start_async_dma();
		}

		int ind = blockIdx.x*COMPUTE_THREADS_PER_CTA + threadIdx.x;
		if (ind<n)
			y[ind] = alpha * res;
	}
	else if (dma_ld_0.owns_this_thread())
	{
#ifdef DYNAMIC
                #pragma unroll 1
		for (int idx=0; idx<n1; idx += (2*VEC_ELMTS))
#else
                for (int idx=0; idx<SGEMV_ITERS; idx++)
#endif
		{
			dma_ld_0.execute_dma(x,buff0);
			x += VEC_ELMTS;
			dma_ld_1.execute_dma(x,buff1);
			x += VEC_ELMTS;
		}	
		dma_ld_0.wait_for_dma_start();
		dma_ld_1.wait_for_dma_start();
	}
	else if (dma_ld_2.owns_this_thread())
	{
                int ind = blockIdx.x*COMPUTE_THREADS_PER_CTA;
                A += ind;
#ifdef DYNAMIC
                #pragma unroll 1
		for (int idx=0; idx<n1; idx += (2*VEC_ELMTS))
#else
                for (int idx=0; idx<SGEMV_ITERS; idx++)
#endif
		{
			dma_ld_2.execute_dma(A,mat0);
			A += (lda*VEC_ELMTS);
			dma_ld_3.execute_dma(A,mat1);
			A += (lda*VEC_ELMTS);
		}
		dma_ld_2.wait_for_dma_start();
		dma_ld_3.wait_for_dma_start();
	}
}

