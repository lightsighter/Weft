#include "hip/hip_runtime.h"
/*
 * Copyright 2015 Stanford University and NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <cmath>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "cudaDMA.h"

#define SIZE_N	        896
#define SIZE_M		SIZE_N

#define DMA_KERNEL			sgemvn_cuda_dma_vec_manual
#define COMPUTE_THREADS_PER_CTA		128	
#define DMA_THREADS_PER_LD		32
#define DMA_LDS				1
#ifndef VEC_ELMTS
#define VEC_ELMTS			512	
#endif

#ifndef SGEMV_ITERS
#define SGEMV_ITERS                     128
#endif

__global__ void
__launch_bounds__(160,1)
sgemvn_cuda_dma_vec_manual(int n, int m, int n1, float alpha, float *A, int lda, float *x, float *y)
{
	__shared__ float buff0[VEC_ELMTS];
	__shared__ float buff1[VEC_ELMTS];


	cudaDMASequential<true,16,4*VEC_ELMTS,DMA_THREADS_PER_LD>
	  dma_ld_0(1,COMPUTE_THREADS_PER_CTA,COMPUTE_THREADS_PER_CTA);
	cudaDMASequential<true,16,4*VEC_ELMTS,DMA_THREADS_PER_LD>
	  dma_ld_1(2,COMPUTE_THREADS_PER_CTA,COMPUTE_THREADS_PER_CTA);

	if (threadIdx.x < COMPUTE_THREADS_PER_CTA)
	{
		dma_ld_0.start_async_dma();	
		dma_ld_1.start_async_dma();
		int ind = blockIdx.x*COMPUTE_THREADS_PER_CTA + threadIdx.x;

		A += ind;

		float res = 0.f;

#ifdef DYNAMIC
                #pragma unroll 1
		for(int i=0; i<n1; i += (VEC_ELMTS*2) )
#else
                for(int i=0; i<SGEMV_ITERS; i++)
#endif
		{
			dma_ld_0.wait_for_dma_finish();
			#pragma unroll
			for(int j=0; j < VEC_ELMTS; j++)
			{
				res+=A[0]*buff0[j];
				A+=lda;
			}
			dma_ld_0.start_async_dma();
			dma_ld_1.wait_for_dma_finish();
			#pragma unroll
			for (int j=0; j < VEC_ELMTS; j++)
			{
				res+=A[0]*buff1[j];
				A+=lda;
			}
			dma_ld_1.start_async_dma();
		}

		if (ind<n)
			y[ind] = alpha * res;
	}
	else if (dma_ld_0.owns_this_thread())
	{
#ifdef DYNAMIC
                #pragma unroll 1
		for (int i=0; i<n1; i += (VEC_ELMTS*2))
#else
                for (int i=0; i<SGEMV_ITERS; i++)
#endif
		{
			dma_ld_0.execute_dma(x,buff0);
			x += VEC_ELMTS;
			dma_ld_1.execute_dma(x,buff1);
			x += VEC_ELMTS;
		}	
		dma_ld_0.wait_for_dma_start();
		dma_ld_1.wait_for_dma_start();
	}
}

